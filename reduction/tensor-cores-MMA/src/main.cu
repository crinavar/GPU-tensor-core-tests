#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <map>
#include <random>
#include <cmath>
#define REAL float
#define TCSIZE 16
#define TCSQ 256
#define PRINTLIMIT 2560
#define WARPSIZE 32
#define DIFF (BSIZE<<3)

#include "tools.cuh"
#include "kernel.cuh"
#include "variants.cuh"


int main(int argc, char **argv){
    // params
    if(argc != 8){
        fprintf(stderr, "run as ./prog dev n factor_ns seed REPEATS dist method\nmethod:\
        \n0 -> warp-shuffle    \
        \n1 -> recurrence\
        \n2 -> chained MMAs\
        \n3 -> split\
        \n4 -> recursive-chained\n\n");
        exit(EXIT_FAILURE);
    }
    int dev = atoi(argv[1]);
    long on = atoi(argv[2]);
    long n = on;
    float factor_ns = atof(argv[3]);
    int seed = atoi(argv[4]);
    int REPEATS = atoi(argv[5]);
    int dist = atoi(argv[6]);
    int method = atoi(argv[7]);

#ifdef DEBUG
    const char* methods[5] = {"WARP-SHUFFLE", "RECURRENCE", "CHAINED MMAs", "SPLIT", "RECURRENCE-CHAINED"};
    const char* disttext[3] = {"Normal Distribution", "Uniform Distribution", "Constant Distribution"};
    printf("\n\
            ***************************\n\
            dev            = %i\n\
            method         = %s\n\
            n              = %i\n\
            factor_ns      = %f\n\
            dist           = %s\n\
            prng_seed      = %i\n\
            REPEATS        = %i\n\
            TCSIZE         = %i\n\
            R              = %i\n\
            BSIZE          = %i\n\
            ***************************\n\n", dev, methods[method], n, factor_ns, disttext[dist], seed, REPEATS, TCSIZE, R, BSIZE);
#endif
    
    // set device
    hipSetDevice(dev);

    // mallocs
    REAL *A, *Ad;
    half *Adh, *outd_recA, *outd_recB;
    float *outd, *out;

    A = (REAL*)malloc(sizeof(REAL)*n);
    out = (float*)malloc(sizeof(float)*1);
    hipMalloc(&Ad, sizeof(REAL)*n);
    hipMalloc(&Adh, sizeof(half)*n);
    hipMalloc(&outd, sizeof(float)*1);
    long smalln = (n + TCSQ-1)/TCSQ;
    //printf("small n = %lu   bs = %i\n", smalln, bs);
    hipMalloc(&outd_recA, sizeof(half)*(smalln));
    hipMalloc(&outd_recB, sizeof(half)*(smalln));

    init_distribution(A, n, seed, dist);
    hipMemcpy(Ad, A, sizeof(REAL)*n, hipMemcpyHostToDevice);
    convertFp32ToFp16 <<< (n + 256 - 1)/256, 256 >>> (Adh, Ad, n);
    hipDeviceSynchronize();
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    #ifdef DEBUG
        printf("%s (BSIZE = %i)\n", methods[method], BSIZE);
    #endif
    hipEventRecord(start);
    switch(method){
        case 0:
            warpshuffle_reduction(Adh, outd, n, REPEATS);
            break;
        case 1:
            recurrence_reduction(Adh, outd, outd_recA, outd_recB, n, REPEATS);
            break;
        case 2:
            chainedMMAs_reduction(Adh, outd, n, REPEATS);
            break;
        case 3:
            split_reduction(Adh, outd, n, factor_ns, REPEATS);
            break;
        case 4:
            recurrence_reduction_chained(Adh, outd, outd_recA, outd_recB, n, REPEATS);
            break;
    }        
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(out, outd, sizeof(float)*1, hipMemcpyDeviceToHost);
    float time = 0.0f;
    hipEventElapsedTime(&time, start, stop);
    double cpusum = gold_reduction(A, n);

    #ifdef DEBUG
        printf("Done:\n\
                Time (GPU):  %f\n\
                GPU Result:  %f\n\
                CPU Result:  %f\n\
                Diff Result: %f\n\
                \% Error   : %f\n\n", time/(REPEATS),(float)*out,cpusum,fabs((float)*out - cpusum),fabs(100.0f*fabs((float)*out - cpusum)/cpusum));
    #else
        printf("%f,%f,%f,%f,%f\n", time/(REPEATS),(float)*out,cpusum,fabs((float)*out - cpusum),fabs(100.0f*fabs((float)*out - cpusum)/cpusum));
    #endif
    free(A);
    free(out);
    hipFree(Ad);
    hipFree(Adh);
    hipFree(outd);
    hipFree(outd_recA);
    hipFree(outd_recB);
    exit(EXIT_SUCCESS);
}

